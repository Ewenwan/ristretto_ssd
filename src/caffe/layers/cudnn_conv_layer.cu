
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_conv_layer.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
// this->conv_iter_  是网络层参数，需要写入protxtx
/*
    // store input  保存初始相应层 的输入
    if ( this->conv_iter_ == 0)// && this->conv_id_ == 2)
    {
      const Dtype* bottom_data_c = bottom[0]->cpu_data();
      int bottom_size = bottom[0]->shape(0) *
        bottom[0]->shape(1) *
        bottom[0]->shape(2) *
        bottom[0]->shape(3);
      LOG(INFO) << "bottom_size: "<< bottom_size;
      //LOG(INFO) << "w1: "<< weight_c[0];

      char filename[12];
      sprintf(filename, "Oinput_%d", this->conv_id_);

      FILE * fp;
      fp = fopen(filename, "wb");
      if (fp !=NULL){
        fwrite(bottom_data_c, 4, bottom_size, fp);
      }
      fclose(fp);
    }
*/

/*
    // store weight and bias  保存 权重和 偏置
    if ( this->conv_iter_ == 0)// && this->conv_id_ == 2)
    {
      const Dtype* weight_c = this->blobs_[0]->cpu_data();
      int w_size = this->blobs_[0]->shape(0) *
        this->blobs_[0]->shape(1) *
        this->blobs_[0]->shape(2) *
        this->blobs_[0]->shape(3);
      LOG(INFO) << "w_size: "<< w_size;
      //LOG(INFO) << "w1: "<< weight_c[0];
    
      char filename[12];
      sprintf(filename, "Oconv_%d", this->conv_id_);
    
      FILE * fp;
      fp = fopen(filename, "wb");
      if (fp !=NULL){
        fwrite(weight_c, 4, w_size, fp);
      }
      fclose(fp);
    }
    if ( this->conv_iter_ == 0 && this->bias_term_)// && this->conv_id_ == 2)
    {
      const Dtype* bias_c = this->blobs_[1]->cpu_data();
      int b_size = this->blobs_[1]->shape(0);
      LOG(INFO) << "b_size: "<< b_size;
      //LOG(INFO) << "w1: "<< weight_c[0];
    
      char filename[12];
      sprintf(filename, "Obias_%d", this->conv_id_);
    
      FILE * fp;
      fp = fopen(filename, "wb");
      if (fp !=NULL){
        fwrite(bias_c, 4, b_size, fp);
      }
      fclose(fp);
    }
*/
      
    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      // Filters.  卷积
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + this->weight_offset_ * g,
            conv_descs_[i],
            fwd_algo_[i], workspace[g], workspace_fwd_sizes_[i],
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.  加上 偏置
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g],
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
      
	/*
    // store output  保存卷积后的输出===================
    if ( this->conv_iter_ == 0)// && this->conv_id_ == 2)
    {
      const Dtype* top_data_c = top[0]->cpu_data();
      int top_size = top[0]->shape(0) *
        top[0]->shape(1) *
        top[0]->shape(2) *
        top[0]->shape(3);
      LOG(INFO) << "top_size: "<< top_size;
      LOG(INFO) << "conv_id: : "<< this->conv_id_;
    
      char filename[12];
      sprintf(filename, "Otop_%d", this->conv_id_);
    
      FILE * fp;
      fp = fopen(filename, "wb");
      if (fp !=NULL){
        fwrite(top_data_c, 4, top_size, fp);
      }
      fclose(fp);
    }
*/
	
	
  }
  
  this->conv_iter_++;// 一开始为0，++
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(
              handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_filter_algo_[i], workspace[1*this->group_ + g],
              workspace_bwd_filter_sizes_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + this->weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(
              handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + this->weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              bwd_data_algo_[i], workspace[2*this->group_ + g],
              workspace_bwd_data_sizes_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
